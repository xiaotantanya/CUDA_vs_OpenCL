#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

#define NUM_ELEMENTS 100000000

int main() {
    // 分配主机内存
    hipfftComplex* h_data = (hipfftComplex*)malloc(sizeof(hipfftComplex) * NUM_ELEMENTS);

    // 分配设备内存
    hipfftComplex* d_data;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * NUM_ELEMENTS);

    // 创建 cuFFT 计划
    hipfftHandle plan;
    hipfftPlan1d(&plan, NUM_ELEMENTS, HIPFFT_C2C, 1);

    // 将输入数据从主机内存复制到设备内存
    hipMemcpy(d_data, h_data, sizeof(hipfftComplex) * NUM_ELEMENTS, hipMemcpyHostToDevice);

    // 执行傅里叶变换
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

    // 将结果从设备内存复制到主机内存
    hipMemcpy(h_data, d_data, sizeof(hipfftComplex) * NUM_ELEMENTS, hipMemcpyDeviceToHost);

    // 释放资源
    hipfftDestroy(plan);
    hipFree(d_data);
    free(h_data);

    return 0;
}
